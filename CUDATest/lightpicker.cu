#include "lightpicker.h"

__device__ LightPicker::LightPicker() {
	intensity = 1.f;
}

__device__ AreaLight* LightPicker::GetLight(Shape* shape, const Color& color, Point* p) const {
	return new AreaLight(shape, color, intensity, p);
}

__device__ void LightPicker::IncreaseIntensity(float step) {
	intensity += step;
}

__device__ void LightPicker::DecreaseIntensity(float step) {
	intensity -= step;
	if (intensity < 0.f) intensity = 0.f;
}