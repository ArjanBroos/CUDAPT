#include "hip/hip_runtime.h"
#include "octree.h"
#include "point.h"
#include "primitive.h"
#include "light.h"
#include "object.h"
#include <iostream>
#include <cmath>

__device__ Node::Node() {
}

__device__ Node::Node(Point bounda, Point boundb, int id) : nObjects(0), object(nullptr), id(id), octant(-1), parent(nullptr)
{
	if(bounda < boundb) {
		bounds[0] = bounda;
		bounds[1] = boundb;
	} else {
		bounds[0] = boundb;
		bounds[1] = bounda;
	}
	for( int i = 0; i < 8; i++) 
		nodes[i] = nullptr;
}

int Node::nextId = 0;

__device__ Node::Node(Point bounda, Point boundb, int octant, Node* parent, int id) : nObjects(0), object(nullptr), id(id), octant(octant), parent(parent)
{

	if(bounda < boundb) {
		bounds[0] = bounda;
		bounds[1] = boundb;
	} else {
		bounds[0] = boundb;
		bounds[1] = bounda;
	}
	for( int i = 0; i < 8; i++) 
		nodes[i] = nullptr;
}

__device__ int Node::Insert(Object* object, int &id)
{
	Point loc = (*object->loc);
	if(loc.x > bounds[1].x - 1 || loc.y > bounds[1].y - 1 || loc.z > bounds[1].z - 1) {
		return -1;
	}
	Node* currentNode = this;
	while( ! ( (fabsf(currentNode->bounds[1].x - currentNode->bounds[0].x - 1) < 1e-5f) 
		&& (fabsf(currentNode->bounds[1].y - currentNode->bounds[0].y - 1) < 1e-5f) 
		&& (fabsf(currentNode->bounds[1].z - currentNode->bounds[0].z - 1) < 1e-5f) ) )
	{
		//Is loc in this bounding box?
		if(loc < currentNode->bounds[0] || loc > currentNode->bounds[1])
			return 0;
		//Find leaf node
		int midx = (int) ((currentNode->bounds[0].x+currentNode->bounds[1].x)/2.0);
		int midy = (int) ((currentNode->bounds[0].y+currentNode->bounds[1].y)/2.0);
		int midz = (int) ((currentNode->bounds[0].z+currentNode->bounds[1].z)/2.0);
		bool east = loc.x >= midx;
		bool north = loc.y >= midy;
		bool top = loc.z >= midz;
		if(east && north && top) {
			if(currentNode->nodes[NET] == nullptr)
				currentNode->nodes[NET] = new Node(Point((float)midx, (float)midy, (float)midz), Point(currentNode->bounds[1].x, currentNode->bounds[1].y, currentNode->bounds[1].z), NET, currentNode, id++);
			currentNode = currentNode->nodes[NET];
		}
		if(!east && north && top) {
			if(currentNode->nodes[NWT] == nullptr)
				currentNode->nodes[NWT] = new Node(Point(currentNode->bounds[0].x, (float)midy, (float)midz), Point((float)midx, currentNode->bounds[1].y, currentNode->bounds[1].z), NWT, currentNode, id++);
			currentNode = currentNode->nodes[NWT];
		}
		if(east && !north && top) {
			if(currentNode->nodes[SET] == nullptr)
				currentNode->nodes[SET] = new Node(Point((float)midx, currentNode->bounds[0].y, (float)midz), Point(currentNode->bounds[1].x, (float)midy, currentNode->bounds[1].z), SET, currentNode, id++);
			currentNode = currentNode->nodes[SET];
		}
		if(!east && !north && top) {
			if(currentNode->nodes[SWT] == nullptr)
				currentNode->nodes[SWT] = new Node(Point(currentNode->bounds[0].x, currentNode->bounds[0].y, (float)midz), Point((float)midx, (float)midy, currentNode->bounds[1].z), SWT, currentNode, id++);
			currentNode = currentNode->nodes[SWT];
		}
		if(east && north && !top) {
			if(currentNode->nodes[NEB] == nullptr)
				currentNode->nodes[NEB] = new Node(Point((float)midx, (float)midy, currentNode->bounds[0].z), Point(currentNode->bounds[1].x, currentNode->bounds[1].y, (float)midz), NEB, currentNode, id++);
			currentNode = currentNode->nodes[NEB];
		}
		if(!east && north && !top) {
			if(currentNode->nodes[NWB] == nullptr)
				currentNode->nodes[NWB] = new Node(Point(currentNode->bounds[0].x, (float)midy, currentNode->bounds[0].z), Point((float)midx, currentNode->bounds[1].y, (float)midz), NWB, currentNode, id++);
			currentNode = currentNode->nodes[NWB];
		}
		if(east && !north && !top) {
			if(currentNode->nodes[SEB] == nullptr)
				currentNode->nodes[SEB] = new Node(Point((float)midx, currentNode->bounds[0].y, currentNode->bounds[0].z), Point(currentNode->bounds[1].x, (float)midy, (float)midz), SEB, currentNode, id++);
			currentNode = currentNode->nodes[SEB];
		}
		if(!east && !north && !top) {
			if(currentNode->nodes[SWB] == nullptr)
				currentNode->nodes[SWB] = new Node(Point(currentNode->bounds[0].x, currentNode->bounds[0].y, currentNode->bounds[0].z), Point((float)midx, (float)midy, (float)midz), SWB, currentNode, id++);
			currentNode = currentNode->nodes[SWB];
		}
	}

	//Smalles node found, try to insert
	if(currentNode->object) {
		return -1;
	}
	currentNode->object = object;
	object->parent = currentNode;

	//Fix number of objects in parents
	while(currentNode->parent != nullptr) {
		currentNode->nObjects++;
		currentNode = currentNode->parent;
	}
	currentNode->nObjects++;
	return 1;
}

__device__ void Node::Remove(Object* object) {
	Node* currentNode, *previousNode;
	currentNode = object->parent;
	//Fix number of objects in parents
	while(currentNode->parent != nullptr) {
		currentNode->nObjects--;
		previousNode = currentNode;
		currentNode = currentNode->parent;
		if(previousNode->nObjects == 0) {
			delete currentNode->nodes[previousNode->octant];
			currentNode->nodes[previousNode->octant] = NULL;
		}
	}
	delete object;
}

__device__ bool Node::Intersect(const Ray &ray, IntRec& intRec) const {
	float temp;
	bool intersect = false;
	Node* current = NextNode(this, ray, intRec.t);
	while(current) {
		if(current->object) {
			if(current->object->Intersect(ray, temp) && temp < intRec.t) {
				intRec.t = temp;
				intersect = true;
				if(current->object->type == PRIMITIVE) {
					intRec.prim = (Primitive*) current->object;
					intRec.light = NULL;
				} else {
					intRec.prim = NULL;
					intRec.light = (Light*) current->object;
				}
			}
		}
		current = NextNode(current, ray, intRec.t);
	}
	return intersect;
}

__device__ Node* Node::NextNode(const Node* current, const Ray &ray, float &closest) const{
	// Return the left most child node that intersects with the ray
	Node* node;
	for(int i = NEB; i <= SET; i++) {
		node = current->nodes[i];
		if(node && node->NodeIntersect(ray) < closest) {
			return node;
		}
	}

	// If it's a leaf node, find the next sibling or ascend and repeat
	while(current->parent) {
		node = current->parent;
		for(int i = current->octant + 1; i <= SET; i++) {
			Node* node2 = node->nodes[i];
			if(node2 && node2->NodeIntersect(ray) < closest) {
				return node2;
			}
		}
		current = current->parent;
	}
	return nullptr;
}

__device__ float Node::NodeIntersect(const Ray &ray) const {
	float tmin, tmax, tminn, tmaxn;

	tmin = (bounds[ray.sign[0]].x - ray.o.x) * ray.inv.x;
	tmax = (bounds[1-ray.sign[0]].x - ray.o.x) * ray.inv.x;
	tminn = (bounds[ray.sign[1]].y - ray.o.y) * ray.inv.y;
	tmaxn = (bounds[1-ray.sign[1]].y - ray.o.y) * ray.inv.y;

	//Compare to previous interval
	if ( (tmin > tmaxn) || (tminn > tmax) )
		return INFINITY;
	if (tminn > tmin)
		tmin = tminn;
	if (tmaxn < tmax)
		tmax = tmaxn;

	tminn = (bounds[ray.sign[2]].z - ray.o.z) * ray.inv.z;
	tmaxn = (bounds[1-ray.sign[2]].z - ray.o.z) * ray.inv.z;
	//Compare to previous interval
	if ( (tmin > tmaxn) || (tminn > tmax) )
		return INFINITY;
	if (tminn > tmin)
		tmin = tminn;
	if (tmaxn < tmax)
		tmax = tmaxn;
	if(tmax < 0)
		return INFINITY;
	if ( (tmin < ray.maxt) && (tmax > ray.mint) && (tmax > 0)) {
		return tmin;
	}
	return INFINITY;
}