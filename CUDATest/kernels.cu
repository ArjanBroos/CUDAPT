#include "hip/hip_runtime.h"
#include "kernels.h"
#include "sphere.h"
#include "intrec.h"
#include "lambertmaterial.h"
#include "arealight.h"
#include "plane.h"
#include "mirrormaterial.h"
#include "box.h"

void LaunchInitRNG(hiprandState* state, unsigned long seed, unsigned width, unsigned height, unsigned tileSize) {
	dim3 grid(width / tileSize, height / tileSize);
	dim3 block(tileSize, tileSize);
	InitRNG<<<grid, block>>>(state, seed, width);
}

__global__ void InitRNG(hiprandState* state, unsigned long seed, unsigned width) {
	const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned i = y * width + x;
	hiprand_init(seed - i, 0, 0, &state[i]);
}

void LaunchInitScene(Scene** pScene) {
	InitScene<<<1,1>>>(pScene);
}

__global__ void InitScene(Scene** pScene) {
	*pScene = new Scene();
	Scene* scene = *pScene;

	Sphere*				sphereShape1	= new Sphere(Point(0.0f, 0.4f, 0.0f), 2.f);
	LambertMaterial*	sphereMat1		= new LambertMaterial(Color(1.f, 0.f, 0.f), 1.f);
	scene->AddPrimitive(new Primitive(sphereShape1, sphereMat1));
	Sphere*				sphereShape2	= new Sphere(Point(3.f, 0.2f, 0.f), 1.f);
	MirrorMaterial*		sphereMat2		= new MirrorMaterial(Color(1.f, 1.f, 1.f), 0.8f);
	scene->AddPrimitive(new Primitive(sphereShape2, sphereMat2));
	Sphere*				sphereShape3	= new Sphere(Point(-2.f, 8.f, -20.f), 6.f);
	MirrorMaterial*		sphereMat3		= new MirrorMaterial(Color(1.f, 1.f, 1.f), 0.9f);
	scene->AddPrimitive(new Primitive(sphereShape3, sphereMat3));

	Box*				boxShape1		= new Box(Point(-1.5f, 0.5f, 3.f), 1.f);
	LambertMaterial*	boxMat1			= new LambertMaterial();
	scene->AddPrimitive(new Primitive(boxShape1, boxMat1));

	Plane*				planeShape1		= new Plane(Point(), Vector(0.f, 1.f, 0.f));
	LambertMaterial*	planeMat1		= new LambertMaterial(Color(1.f, 1.f, 0.3f), 1.f);
	scene->AddPrimitive(new Primitive(planeShape1, planeMat1));

	Sphere*				lightShape1		= new Sphere(Point(-2.f, 3.f, 1.f), 1.5f);
	scene->AddLight(new AreaLight(lightShape1));
	Sphere*				lightShape2		= new Sphere(Point(5.f, 1.f, -3.f), 0.8f);
	scene->AddLight(new AreaLight(lightShape2, Color(1.f, 0.5f, 0.5f), 1.5f));
	Box*				lightShape3		= new Box(Point(3.f, 0.f, 5.f), 0.8f);
	scene->AddLight(new AreaLight(lightShape3, Color(1.f, 1.f, 1.f), 2.f));
}

void LaunchInitResult(Color* result, unsigned width, unsigned height, unsigned tileSize) {
	dim3 grid(width / tileSize, height / tileSize);
	dim3 block(tileSize, tileSize);
	InitResult<<<grid, block>>>(result, width);
}

__global__ void InitResult(Color* result, unsigned width) {
	const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned i = y * width + x;
	result[i] = Color();
}

void LaunchTraceRays(const Camera* cam, const Scene* scene, Color* result, hiprandState* rng, unsigned width, unsigned height, unsigned tileSize) {
	dim3 grid(width / tileSize, height / tileSize);
	dim3 block(tileSize, tileSize);
	TraceRays<<<grid, block>>>(cam, scene, result, rng, width);
}

__global__ void TraceRays(const Camera* cam, const Scene* scene, Color* result, hiprandState* rng, unsigned width) {
	const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned i = y * width + x;

	Ray ray = cam->GetJitteredRay(x, y, rng);
	const unsigned maxDepth = 4;
	IntRec intRec;
	Color color(1.f, 1.f, 1.f);

	for (unsigned depth = 0; depth <= maxDepth; depth++) {
		if (depth == maxDepth) {
			color *= Color();
			break;
		}

		if (!scene->Intersect(ray, intRec)) {
			color *= Color(0.2f, 0.2f, 0.3f);
			break;
		}

		if (intRec.light) {
			color *= intRec.light->Le();
			break;
		}

		const Material* mat = intRec.prim->GetMaterial();
		const Shape*	shape = intRec.prim->GetShape();
		const Point		p = ray(intRec.t);
		const Vector	n = shape->GetNormal(p);

		const Vector in = ray.d;
		const Vector out = mat->GetSample(in, n, &rng[i]);
		ray = Ray(p, out);

		color *= mat->GetColor();
		color *= mat->GetMultiplier(in, out, n);
	}

	result[i] += color;
}

void LaunchConvert(const Color* result, unsigned char* pixelData, unsigned iteration, unsigned width, unsigned height, unsigned tileSize) {
	dim3 grid(width / tileSize, height / tileSize);
	dim3 block(tileSize, tileSize);
	Convert<<<grid, block>>>(result, pixelData, iteration, width);
}

__global__ void Convert(const Color* result, unsigned char* pixelData, unsigned iteration, unsigned width) {
	const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned i = y * width + x;
	const unsigned pdi = i*4;

	const float r = (result[i].r / iteration) * 255;
	const float g = (result[i].g / iteration) * 255;
	const float b = (result[i].b / iteration) * 255;

	pixelData[pdi]		= Clamp255(r);
	pixelData[pdi+1]	= Clamp255(g);
	pixelData[pdi+2]	= Clamp255(b);
	pixelData[pdi+3]	= 255;
}

void LaunchDestroyScene(Scene* scene) {
	DestroyScene<<<1,1>>>(scene);
}

__global__ void DestroyScene(Scene* scene) {
	delete scene;
	scene = NULL;
}

__device__ unsigned char Clamp255(float s) {
	if (s < 0.f) s = 0.f;
	if (s > 255.f) s = 255.f;
	return (unsigned char)s;
}