#include "hip/hip_runtime.h"
#include "lambertmaterial.h"
#include "math.h"

// Initializes a white material with an albedo of 1
__device__ LambertMaterial::LambertMaterial() : color(Color(1.f, 1.f, 1.f)), albedo(1.f) {
}

// Initializes a material with given color and albedo
__device__ LambertMaterial::LambertMaterial(const Color& color, float albedo) : color(color), albedo(albedo) {
}

// Returns the color of this material
__device__ Color LambertMaterial::GetColor() const {
	return color;
}

// Bidirectional Reflectance Distribution Function
__device__ float LambertMaterial::GetBRDF(const Vector& in, const Vector& out, const Vector& normal) const {
	return albedo / M_PI;
}

// Probability Density Function for cosine-weighted hemisphere sampling
__device__ float LambertMaterial::GetPDF(const Vector& in, const Vector& out, const Vector& normal) const {
	return Dot(out, normal) / M_PI;
}

// Cosine weighted sampling on the unit hemisphere
__device__ Vector LambertMaterial::GetSample(const Vector& in, const Vector& normal, hiprandState* rng) const {
	const Vector u = Normalize(Vector(normal.y, normal.z - normal.x, -normal.y)); // A vector perpendicular to the normal
	const Vector v = Cross(u, normal); // Another vector perpendicular to both u and the normal

	const float u1 = hiprand_uniform(rng);
	const float u2 = hiprand_uniform(rng);

	const float r = sqrtf(u1);
	const float phi = u2 * 2.f * (float)M_PI;

	const float x = r * cosf(phi);
	const float y = sqrtf(fmaxf(0.f, 1.f - u1));
	const float z = r * sinf(phi);

	return u * x + normal * y + v * z;
}

// Returns the factor between incoming and outgoing radiance along given rays
__device__ float LambertMaterial::GetMultiplier(const Vector& in, const Vector& out, const Vector& normal) const {
	return albedo;
}